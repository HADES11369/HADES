#include <hip/hip_runtime.h>
#include <torch/script.h>
#include "../utils.h"
#include "ops.h"

namespace distemb {
namespace cuda {

void PinTensor(torch::Tensor tensor) {
  CUDA_CALL(hipHostRegister(const_cast<void *>(tensor.storage().data()),
                             tensor.numel() * tensor.element_size(),
                             hipHostRegisterDefault));
}

void UnpinTensor(torch::Tensor tensor) {
  CUDA_CALL(hipHostUnregister(const_cast<void *>(tensor.storage().data())));
}

void RegisterPtr(int64_t ptr, int64_t size) {
  CUDA_CALL(hipHostRegister(reinterpret_cast<void *>(ptr), size,
                             hipHostRegisterDefault));
}

void UnregisterPtr(int64_t ptr) {
  CUDA_CALL(hipHostUnregister(reinterpret_cast<void *>(ptr)));
}

}  // namespace cuda
}  // namespace distemb